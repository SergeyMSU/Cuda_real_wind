#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <math.h>
#include <vector>
#include <string>
#include <cmath>

#define L 1.0
#define minut 0.000150403  // 1 ������ � ������������ �������.
#define hour 0.00902419      // 1 ��� � ������������ �������.
#define day 0.216581      // 1 ���� � ������������ �������.
#define dx 0.00125  //0.00025  // 0.0001         // ������ ������ � �.�.
#define THREADS_PER_BLOCK 256
#define Omni_ 20000 //9650


#define ga (5.0/3.0)          // ���������� ��������
#define ggg (5.0/3.0)
#define kv(x) ((x)*(x))
#define kvv(x,y,z)  (kv(x) + kv(y) + kv(z))
#define U8(ro, p, u, v, w, bx, by, bz)  (p / (ggg - 1.0) + 0.5 * ro * kvv(u,v,w) + kvv(bx,by,bz) / cpi8)
#define skk(u,v,w,bx,by,bz) (u*bx + v*by + w*bz)
#define g1 (ga - 1.0)
#define gg1 (ga - 1.0)
#define g2 (ga + 1.0)
#define gg2 (ga + 1.0)
#define gp ((g2/ga)/2.0)
#define gm ((g1/ga)/2.0)
#define gga ga
#define Omega 0.0
#define eps 10e-10
#define eps8 10e-8
#define kurant  0.2 //0.2
#define pi 3.14159265358979323846
#define PI 3.14159265358979323846
#define cpi4 12.56637061435917295384
#define cpi8 25.13274122871834590768
#define spi4 __dsqrt_rn(cpi4)
#define epsb 1e-6
#define eps_p 1e-6
#define eps_d 1e-3
#define krit 0.2  // 0.2



#define a_2 0.162294  // 0.10263
#define sigma(x) (kv(1.0 - a_2 * log(x)))

#define c_H 0.0391451
#define v_H -0.0704
#define Kn 0.392914    // ������������ ����������� ����� ����������� �������
#define n_H 0.00714286   // 0.05  ������������ ������������ ������ ��������
#define r0 2.7268   // ������������ r0 � ��������� �������� � ����������

using namespace std;

__device__ double linear(const double& x1, const double& t1, const double& x2, const double& t2, const double& y);
__device__ double HLLC_2d_Korolkov_b_s(const double& ro_L, const double& Q_L, const double& p_L, const double& v1_L, const double& v2_L,//
    const double& pp_L, const double& ro_R, const double& Q_R, const double& p_R, const double& v1_R, const double& v2_R, const double& pp_R, const double& W, //
    double* P, double& PQ, const double& n1, const double& n2, const double& rad, double& RO_p, double& P_p, int metod = 1, bool nul_potok = false);
__device__ double minmod(const double& x, const double& y);
__device__ double sign(const double& x);

hipError_t addWithCuda(double* ro, double* p, double* u, int& N);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void funk_time(double* T, double* T_do, double* TT)
{
    *T_do = *T;
    *TT = *TT + *T_do;
    *T = 0.1 * day;// 0.5 * minut;
    return;
}

__device__ double HLLC_2d_Korolkov_b_s(const double& ro_L, const double& Q_L, const double& p_L, const double& v1_L, const double& v2_L,//
    const double& pp_L, const double& ro_R, const double& Q_R, const double& p_R, const double& v1_R, const double& v2_R, const double& pp_R, const double& W, //
    double* P, double& PQ, const double& n1, const double& n2, const double& rad, double& RO_p, double& P_p, int metod, bool nul_potok)
    // BestSeries
    // ������ ���������� 2� ���������
    //
    //  �����:
    // P[1]       // ��������
    // P[2]
    // P[0]       // �����
    // P[3]       // �������
{
    double t1 = n2;
    double t2 = -n1;

    double rop_L = ro_L;// -Q_L;
    double rop_R = ro_R;// -Q_R;

    double u1, v1, u2, v2;
    u1 = v1_L * n1 + v2_L * n2;
    v1 = v1_L * t1 + v2_L * t2;
    u2 = v1_R * n1 + v2_R * n2;
    v2 = v1_R * t1 + v2_R * t2;

    double sqrtroL = sqrt(ro_L);
    double sqrtroR = sqrt(ro_R);
    double cL = sqrt(ggg * p_L / ro_L);
    double cR = sqrt(ggg * p_R / ro_R);


    double uu_L = (kv(v1_L) + kv(v2_L)) / 2.0;
    double uu_R = (kv(v1_R) + kv(v2_R)) / 2.0;



    double SL = min(u1, u2) - max(cL, cR);
    double SR = max(u1, u2) + max(cL, cR);
    double suR = (SR - u2);
    double suL = (SL - u1);

    double SM = (suR * ro_R * u2 - suL * ro_L * u1 - p_R + p_L) //
        / (suR * ro_R - suL * ro_L);

    //double PTT = (suR * ro_R * p_L - suL * ro_L * p_R + ro_L * ro_R * suR * suL * (u2 - u1)) / (suR * ro_R - suL * ro_L);

    double UU = max(fabs(SL), fabs(SR));
    double time = kurant * rad / UU;

    double FL[6], FR[6], UL[6], UR[6];

    double e1 = p_L / g1 + ro_L * uu_L;
    double e2 = p_R / g1 + ro_R * uu_R;
    double ep1 = pp_L / g1 + rop_L * uu_L;
    double ep2 = pp_R / g1 + rop_R * uu_R;


    FL[0] = ro_L * u1;
    FL[1] = ro_L * u1 * u1 + p_L;
    FL[2] = ro_L * u1 * v1;
    FL[3] = (e1 + p_L) * u1;
    FL[4] = Q_L * u1;
    FL[5] = (ep1 + pp_L) * u1;

    FR[0] = ro_R * u2;
    FR[1] = ro_R * u2 * u2 + p_R;
    FR[2] = ro_R * u2 * v2;
    FR[3] = (e2 + p_R) * u2;
    FR[4] = Q_R * u2;
    FR[5] = (ep2 + pp_R) * u2;


    UL[0] = ro_L;
    UL[1] = ro_L * u1;
    UL[2] = ro_L * v1;
    UL[3] = e1;
    UL[4] = Q_L;
    UL[5] = ep1;

    UR[0] = ro_R;
    UR[1] = ro_R * u2;
    UR[2] = ro_R * v2;
    UR[3] = e2;
    UR[4] = Q_R;
    UR[5] = ep2;

    if (SL >= W)
    {
        P[1] = n1 * (FL[1] - W * UL[1]) + t1 * (FL[2] - W * UL[2]);     // ��������
        P[2] = n2 * (FL[1] - W * UL[1]) + t2 * (FL[2] - W * UL[2]);
        P[0] = FL[0] - W * UL[0];                       // �����
        P[3] = FL[3] - W * UL[3];                       // �������
        PQ = FL[4] - W * UL[4];
        P[4] = FL[5] - W * UL[5];                       // ��� ������� ��������
        RO_p = rop_L;
        P_p = pp_L;
        return time;
    }

    if (SR <= W)
    {
        P[1] = n1 * (FR[1] - W * UR[1]) + t1 * (FR[2] - W * UR[2]);     // ��������
        P[2] = n2 * (FR[1] - W * UR[1]) + t2 * (FR[2] - W * UR[2]);
        P[0] = FR[0] - W * UR[0];                       // �����
        P[3] = FR[3] - W * UR[3];                       // �������
        PQ = FR[4] - W * UR[4];
        P[4] = FR[5] - W * UR[5];
        RO_p = rop_R;
        P_p = pp_R;
        return time;
    }

    //printf("TUT\n");
    double ro_LL = ro_L * (SL - u1) / (SL - SM);
    double ro_RR = ro_R * (SR - u2) / (SR - SM);
    double Q_LL = Q_L * (SL - u1) / (SL - SM);
    double Q_RR = Q_R * (SR - u2) / (SR - SM);
    double rop_LL = rop_L * (SL - u1) / (SL - SM);
    double rop_RR = rop_R * (SR - u2) / (SR - SM);


    double UZ0 = (SR * UR[0] - SL * UL[0] + FL[0] - FR[0]) / (SR - SL);
    double UZ1 = (SR * UR[1] - SL * UL[1] + FL[1] - FR[1]) / (SR - SL);
    double UZ2 = (SR * UR[2] - SL * UL[2] + FL[2] - FR[2]) / (SR - SL);
    double UZ3 = (SR * UR[3] - SL * UL[3] + FL[3] - FR[3]) / (SR - SL);
    double UZ4 = (SR * UR[4] - SL * UL[4] + FL[4] - FR[4]) / (SR - SL);
    double UZ5 = (SR * UR[5] - SL * UL[5] + FL[5] - FR[5]) / (SR - SL);
    double vzL, vzR, vLL, vRR, ppLR, ee1, ee2, eep1, eep2;

    // ��� ���������� �� �������� ��� ������ � ������������ ��������
    //if (metod == 0)
    //{
    //    double  PO[5];
    //    for (int i = 0; i < 6; i++)
    //    {
    //        PO[i] = (SR * FL[i] - SL * FR[i] + SR * SL * (UR[i] - UL[i])) / (SR - SL);
    //    }

    //    P[1] = n1 * (PO[1] - W * UZ1) + t1 * (PO[2] - W * UZ2);     // ��������
    //    P[2] = n2 * (PO[1] - W * UZ1) + t2 * (PO[2] - W * UZ2);
    //    P[0] = PO[0] - W * UZ0;                       // �����
    //    P[3] = PO[3] - W * UZ3;                       // �������
    //    PQ = PO[4] - W * UZ4;
    //    P[4] = PO[5] - W * UZ5;
    //    return time;
    //}


    double suRm = suR / (SR - SM);
    double suLm = suL / (SL - SM);
    double rzR = ro_R * suRm;
    double rzL = ro_L * suLm;

    double ptzR = p_R + ro_R * suR * (SM - u2);
    double ptzL = p_L + ro_L * suL * (SM - u1);
    double ptz = (ptzR + ptzL) / 2.0;

    double ptzpR = pp_R + rop_R * suR * (SM - u2);
    double ptzpL = pp_L + rop_L * suL * (SM - u1);
    double ptzp = (ptzpR + ptzpL) / 2.0;

    P_p = ptzp;
    /*if( fabs(v1 - v2) > 0.1)
    {
        vLL = v1;
        vRR = v2;
    }
    else
    {
        vRR = UZ2 / UZ0;
        vLL = vRR;
    }*/


    if (nul_potok == true)   // ����� �����������
    {
        vRR = UZ2 / UZ0;
        vLL = vRR;
    }
    else
    {
        vLL = v1;
        vRR = v2;
    }



    ee2 = e2 * suRm + (ptz * SM - p_R * u2) / (SR - SM);
    ee1 = e1 * suLm + (ptz * SM - p_L * u1) / (SL - SM);
    eep2 = ep2 * suRm + (ptzp * SM - pp_R * u2) / (SR - SM);
    eep1 = ep1 * suLm + (ptzp * SM - pp_L * u1) / (SL - SM);


    double  ULL[6], URR[6], PO[6];
    ULL[0] = ro_LL;
    ULL[1] = ro_LL * SM;
    ULL[2] = ro_LL * vLL;
    ULL[3] = ee1;
    ULL[4] = Q_LL;
    ULL[5] = eep1;

    URR[0] = ro_RR;
    URR[1] = ro_RR * SM;
    URR[2] = ro_RR * vRR;
    URR[3] = ee2;
    URR[4] = Q_RR;
    URR[5] = eep2;

    if (SL < W && SM >= W)
    {
        for (int i = 0; i < 6; i++)
        {
            PO[i] = FL[i] + SL * ULL[i] - SL * UL[i] - W * ULL[i];
        }
        RO_p = rop_LL;
    }
    else if (SR > W && SM < W)
    {
        for (int i = 0; i < 6; i++)
        {
            PO[i] = FR[i] + SR * URR[i] - SR * UR[i] - W * URR[i];
        }
        RO_p = rop_RR;
    }

    P[1] = n1 * PO[1] + t1 * PO[2];     // ��������
    P[2] = n2 * PO[1] + t2 * PO[2];
    P[0] = PO[0];                       // �����
    P[3] = PO[3];                       // �������
    PQ = PO[4];
    P[4] = PO[5];
    return time;
}

__device__ double HLLDQ_Korolkov(const double& ro_L, const double& Q_L, const double& p_L, const double& v1_L, const double& v2_L, const double& v3_L,//
    const double& Bx_L, const double& By_L, const double& Bz_L, const double& ro_R, const double& Q_R, const double& p_R, const double& v1_R, const double& v2_R, const double& v3_R,//
    const double& Bx_R, const double& By_R, const double& Bz_R, double* P, double& PQ, const double& n1, const double& n2, const double& n3, double& rad, int metod)
{// �� ��������, ���� �������� ����� �� �������
 // ������� ����� ��������� �� ���� ��������� ! (����� ����� ������� ������������)

    double bx_L = Bx_L / spi4;
    double by_L = By_L / spi4;
    double bz_L = Bz_L / spi4;

    double bx_R = Bx_R / spi4;
    double by_R = By_R / spi4;
    double bz_R = Bz_R / spi4;

    double t1 = 0.0;
    double t2 = 0.0;
    double t3 = 0.0;

    double m1 = 0.0;
    double m2 = 0.0;
    double m3 = 0.0;

    if (n1 > 0.1)
    {
        t2 = 1.0;
        m3 = 1.0;
    }
    else if (n2 > 0.1)
    {
        t3 = 1.0;
        m1 = 1.0;
    }
    else if (n3 > 0.1)
    {
        t1 = 1.0;
        m2 = 1.0;
    }
    else if (n1 < -0.1)
    {
        t3 = -1.0;
        m2 = -1.0;
    }
    else if (n2 < -0.1)
    {
        t1 = -1.0;
        m3 = -1.0;
    }
    else if (n3 < -0.1)
    {
        t1 = -1.0;
        m2 = -1.0;
    }
    else
    {
        printf("EROROR 1421  normal_error\n");
    }


    double u1, v1, w1, u2, v2, w2;
    u1 = v1_L * n1 + v2_L * n2 + v3_L * n3;
    v1 = v1_L * t1 + v2_L * t2 + v3_L * t3;
    w1 = v1_L * m1 + v2_L * m2 + v3_L * m3;
    u2 = v1_R * n1 + v2_R * n2 + v3_R * n3;
    v2 = v1_R * t1 + v2_R * t2 + v3_R * t3;
    w2 = v1_R * m1 + v2_R * m2 + v3_R * m3;

    double bn1, bt1, bm1, bn2, bt2, bm2;
    bn1 = bx_L * n1 + by_L * n2 + bz_L * n3;
    bt1 = bx_L * t1 + by_L * t2 + bz_L * t3;
    bm1 = bx_L * m1 + by_L * m2 + bz_L * m3;
    bn2 = bx_R * n1 + by_R * n2 + bz_R * n3;
    bt2 = bx_R * t1 + by_R * t2 + bz_R * t3;
    bm2 = bx_R * m1 + by_R * m2 + bz_R * m3;

    //cout << " = " << bt2 * bt2 + bm2 * bm2 << endl;

    double sqrtroL = sqrt(ro_L);
    double sqrtroR = sqrt(ro_R);
    double ca_L = bn1 / sqrtroL;
    double ca_R = bn2 / sqrtroR;
    double cL = sqrt(ggg * p_L / ro_L);
    double cR = sqrt(ggg * p_R / ro_R);

    double bb_L = kv(bx_L) + kv(by_L) + kv(bz_L);
    double bb_R = kv(bx_R) + kv(by_R) + kv(bz_R);

    double aL = (kv(bx_L) + kv(by_L) + kv(bz_L)) / ro_L;
    double aR = (kv(bx_L) + kv(by_L) + kv(bz_L)) / ro_L;

    double uu_L = (kv(v1_L) + kv(v2_L) + kv(v3_L)) / 2.0;
    double uu_R = (kv(v1_R) + kv(v2_R) + kv(v3_R)) / 2.0;

    double cfL = sqrt((ggg * p_L + bb_L + //
        sqrt(kv(ggg * p_L + bb_L) - 4.0 * ggg * p_L * kv(bn1))) / (2.0 * ro_L));
    double cfR = sqrt((ggg * p_R + bb_R + //
        sqrt(kv(ggg * p_R + bb_R) - 4.0 * ggg * p_R * kv(bn2))) / (2.0 * ro_R));


    double SL = min(u1, u2) - max(cfL, cfR);
    double SR = max(u1, u2) + max(cfL, cfR);

    double pTL = p_L + bb_L / 2.0;
    double pTR = p_R + bb_R / 2.0;

    double suR = (SR - u2);
    double suL = (SL - u1);

    double SM = (suR * ro_R * u2 - suL * ro_L * u1 - pTR + pTL) //
        / (suR * ro_R - suL * ro_L);

    double PTT = (suR * ro_R * pTL - suL * ro_L * pTR + ro_L * ro_R * suR * suL * (u2 - u1))//
        / (suR * ro_R - suL * ro_L);

    double UU = max(fabs(SL), fabs(SR));
    double time = krit * rad / UU;

    double FL[9], FR[9], UL[9], UR[9];

    double e1 = p_L / g1 + ro_L * uu_L + bb_L / 2.0;
    double e2 = p_R / g1 + ro_R * uu_R + bb_R / 2.0;


    FL[0] = ro_L * u1;
    FL[1] = ro_L * u1 * u1 + pTL - kv(bn1);
    FL[2] = ro_L * u1 * v1 - bn1 * bt1;
    FL[3] = ro_L * u1 * w1 - bn1 * bm1;
    FL[4] = (e1 + pTL) * u1 - bn1 * (u1 * bn1 + v1 * bt1 + w1 * bm1);
    //cout << uu_L << endl;
    FL[5] = 0.0;
    FL[6] = u1 * bt1 - v1 * bn1;
    FL[7] = u1 * bm1 - w1 * bn1;
    FL[8] = Q_L * u1;

    FR[0] = ro_R * u2;
    FR[1] = ro_R * u2 * u2 + pTR - kv(bn2);
    FR[2] = ro_R * u2 * v2 - bn2 * bt2;
    FR[3] = ro_R * u2 * w2 - bn2 * bm2;
    FR[4] = (e2 + pTR) * u2 - bn2 * (u2 * bn2 + v2 * bt2 + w2 * bm2);
    FR[5] = 0.0;
    FR[6] = u2 * bt2 - v2 * bn2;
    FR[7] = u2 * bm2 - w2 * bn2;
    FR[8] = Q_R * u2;

    UL[0] = ro_L;
    UL[1] = ro_L * u1;
    UL[2] = ro_L * v1;
    UL[3] = ro_L * w1;
    UL[4] = e1;
    UL[5] = bn1;
    UL[6] = bt1;
    UL[7] = bm1;
    UL[8] = Q_L;

    UR[0] = ro_R;
    UR[1] = ro_R * u2;
    UR[2] = ro_R * v2;
    UR[3] = ro_R * w2;
    UR[4] = e2;
    UR[5] = bn2;
    UR[6] = bt2;
    UR[7] = bm2;
    UR[8] = Q_R;

    double bn = (SR * UR[5] - SL * UL[5] + FL[5] - FR[5]) / (SR - SL);
    double bt = (SR * UR[6] - SL * UL[6] + FL[6] - FR[6]) / (SR - SL);
    double bm = (SR * UR[7] - SL * UL[7] + FL[7] - FR[7]) / (SR - SL);
    double bbn = bn * bn;

    double ro_LL = ro_L * (SL - u1) / (SL - SM);
    double ro_RR = ro_R * (SR - u2) / (SR - SM);
    double Q_LL = Q_L * (SL - u1) / (SL - SM);
    double Q_RR = Q_R * (SR - u2) / (SR - SM);

    if (metod == 2)   // HLLC  + mgd
    {
        double sbv1 = u1 * bn1 + v1 * bt1 + w1 * bm1;
        double sbv2 = u2 * bn2 + v2 * bt2 + w2 * bm2;

        double UZ0 = (SR * UR[0] - SL * UL[0] + FL[0] - FR[0]) / (SR - SL);
        double UZ1 = (SR * UR[1] - SL * UL[1] + FL[1] - FR[1]) / (SR - SL);
        double UZ2 = (SR * UR[2] - SL * UL[2] + FL[2] - FR[2]) / (SR - SL);
        double UZ3 = (SR * UR[3] - SL * UL[3] + FL[3] - FR[3]) / (SR - SL);
        double UZ4 = (SR * UR[4] - SL * UL[4] + FL[4] - FR[4]) / (SR - SL);
        double vzL, vzR, vLL, wLL, vRR, wRR, ppLR, btt1, bmm1, btt2, bmm2, ee1, ee2;


        double suRm = suR / (SR - SM);
        double suLm = suL / (SL - SM);
        double rzR = ro_R * suRm;
        double rzL = ro_L * suLm;

        double ptzR = pTR + ro_R * suR * (SM - u2);
        double ptzL = pTL + ro_L * suL * (SM - u1);
        double ptz = (ptzR + ptzL) / 2.0;


        vRR = UZ2 / UZ0;
        wRR = UZ3 / UZ0;
        vLL = vRR;
        wLL = wRR;

        /*vRR = v2 + bn * (bt2 - bt) / suR / ro_R;
        wRR = w2 + bn * (bm2 - bm) / suR / ro_R;
        vLL = v1 + bn * (bt1 - bt) / suL / ro_L;
        wLL = w1 + bn * (bm1 - bm) / suL / ro_L;*/

        btt2 = bt;
        bmm2 = bm;
        btt1 = btt2;
        bmm1 = bmm2;

        double sbvz = (bn * UZ1 + bt * UZ2 + bm * UZ3) / UZ0;

        ee2 = e2 * suRm + (ptz * SM - pTR * u2 + bn * (sbv2 - sbvz)) / (SR - SM);
        ee1 = e1 * suLm + (ptz * SM - pTL * u1 + bn * (sbv1 - sbvz)) / (SL - SM);

        /*if (fabs(bn) < 0.000001 )
        {
            vRR = v2;
            wRR = w2;
            vLL = v1;
            wLL = w1;
            btt2 = bt2 * suRm;
            bmm2 = bm2 * suRm;
            btt1 = bt1 * suLm;
            bmm1 = bm1 * suLm;
        }*/

        /*ppLR = (pTL + ro_L * (SL - u1) * (SM - u1) + pTR + ro_R * (SR - u2) * (SM - u2)) / 2.0;

        if (fabs(bn) < 0.000001)
        {
            vLL = v1;
            wLL = w1;
            vRR = v2;
            wRR = w2;

            btt1 = bt1 * (SL - u1) / (SL - SM);
            btt2 = bt2 * (SR - u2) / (SR - SM);

            bmm1 = bm1 * (SL - u1) / (SL - SM);
            bmm2 = bm2 * (SR - u2) / (SR - SM);

            ee1 = ((SL - u1) * e1 - pTL * u1 + ppLR * SM) / (SL - SM);
            ee2 = ((SR - u2) * e2 - pTL * u2 + ppLR * SM) / (SR - SM);
        }
        else
        {
            btt2 = btt1 = (SR * UR[6] - SL * UL[6] + FL[6] - FR[6]) / (SR - SL);
            bmm2 = bmm1 = (SR * UR[7] - SL * UL[7] + FL[7] - FR[7]) / (SR - SL);
            vLL = v1 + bn * (bt1 - btt1) / (ro_L * (SL - u1));
            vRR = v2 + bn * (bt2 - btt2) / (ro_R * (SR - u2));

            wLL = w1 + bn * (bm1 - bmm1) / (ro_L * (SL - u1));
            wRR = w2 + bn * (bm2 - bmm2) / (ro_R * (SR - u2));

            double sks1 = u1 * bn1 + v1 * bt1 + w1 * bm1 - SM * bn - vLL * btt1 - wLL * bmm1;
            double sks2 = u2 * bn2 + v2 * bt2 + w2 * bm2 - SM * bn - vRR * btt2 - wRR * bmm2;

            ee1 = ((SL - u1) * e1 - pTL * u1 + ppLR * SM + bn * sks1) / (SL - SM);
            ee2 = ((SR - u2) * e2 - pTR * u2 + ppLR * SM + bn * sks2) / (SR - SM);
        }*/


        double  ULL[9], URR[9], PO[9];
        ULL[0] = ro_LL;
        ULL[1] = ro_LL * SM;
        ULL[2] = ro_LL * vLL;
        ULL[3] = ro_LL * wLL;
        ULL[4] = ee1;
        ULL[5] = bn;
        ULL[6] = btt1;
        ULL[7] = bmm1;
        ULL[8] = Q_LL;

        URR[0] = ro_RR;
        URR[1] = ro_RR * SM;
        URR[2] = ro_RR * vRR;
        URR[3] = ro_RR * wRR;
        URR[4] = ee2;
        URR[5] = bn;
        URR[6] = btt2;
        URR[7] = bmm2;
        URR[8] = Q_RR;

        if (SL >= 0.0)
        {
            for (int i = 0; i < 9; i++)
            {
                PO[i] = FL[i];
            }
        }
        else if (SL < 0.0 && SM >= 0.0)
        {
            for (int i = 0; i < 9; i++)
            {
                PO[i] = FL[i] + SL * ULL[i] - SL * UL[i];
            }
        }
        else if (SR > 0.0 && SM < 0.0)
        {
            for (int i = 0; i < 9; i++)
            {
                PO[i] = FR[i] + SR * URR[i] - SR * UR[i];
            }
        }
        else if (SR <= 0.0)
        {
            for (int i = 0; i < 9; i++)
            {
                PO[i] = FR[i];
            }
        }



        double SN = max(fabs(SL), fabs(SR));

        PO[5] = -SN * (bn2 - bn1);

        P[1] = n1 * PO[1] + t1 * PO[2] + m1 * PO[3];
        P[2] = n2 * PO[1] + t2 * PO[2] + m2 * PO[3];
        P[3] = n3 * PO[1] + t3 * PO[2] + m3 * PO[3];
        P[5] = spi4 * (n1 * PO[5] + t1 * PO[6] + m1 * PO[7]);
        P[6] = spi4 * (n2 * PO[5] + t2 * PO[6] + m2 * PO[7]);
        P[7] = spi4 * (n3 * PO[5] + t3 * PO[6] + m3 * PO[7]);
        P[0] = PO[0];
        P[4] = PO[4];
        PQ = PO[8];

        double SWAP = P[4];
        P[4] = P[5];
        P[5] = P[6];
        P[6] = P[7];
        P[7] = SWAP;
        return time;

    }
    else if (metod == 3)  // HLLD
    {

        double ttL = ro_L * suL * (SL - SM) - bbn;
        double ttR = ro_R * suR * (SR - SM) - bbn;

        double vLL, wLL, vRR, wRR, btt1, bmm1, btt2, bmm2;

        if (fabs(ttL) >= 0.000001)
        {
            vLL = v1 - bn * bt1 * (SM - u1) / ttL;
            wLL = w1 - bn * bm1 * (SM - u1) / ttL;
            btt1 = bt1 * (ro_L * suL * suL - bbn) / ttL;
            bmm1 = bm1 * (ro_L * suL * suL - bbn) / ttL;
        }
        else
        {
            vLL = v1;
            wLL = w1;
            btt1 = 0.0;
            bmm1 = 0.0;
        }

        if (fabs(ttR) >= 0.000001)
        {
            vRR = v2 - bn * bt2 * (SM - u2) / ttR;
            wRR = w2 - bn * bm2 * (SM - u2) / ttR;
            btt2 = bt2 * (ro_R * suR * suR - bbn) / ttR;
            bmm2 = bm2 * (ro_R * suR * suR - bbn) / ttR;
            //cout << "tbr = " << (ro_R * suR * suR - bbn) / ttR << endl;
            //cout << "bt2 = " << bt2 << endl;
        }
        else
        {
            vRR = v2;
            wRR = w2;
            btt2 = 0.0;
            bmm2 = 0.0;
        }

        double eLL = (e1 * suL + PTT * SM - pTL * u1 + bn * //
            ((u1 * bn1 + v1 * bt1 + w1 * bm1) - (SM * bn + vLL * btt1 + wLL * bmm1))) //
            / (SL - SM);
        double eRR = (e2 * suR + PTT * SM - pTR * u2 + bn * //
            ((u2 * bn2 + v2 * bt2 + w2 * bm2) - (SM * bn + vRR * btt2 + wRR * bmm2))) //
            / (SR - SM);

        double sqrtroLL = sqrt(ro_LL);
        double sqrtroRR = sqrt(ro_RR);
        double SLL = SM - fabs(bn) / sqrtroLL;
        double SRR = SM + fabs(bn) / sqrtroRR;

        double idbn = 1.0;
        if (fabs(bn) > 0.0001)
        {
            idbn = 1.0 * sign(bn);
        }
        else
        {
            idbn = 0.0;
            SLL = SM;
            SRR = SM;
        }

        double vLLL = (sqrtroLL * vLL + sqrtroRR * vRR + //
            idbn * (btt2 - btt1)) / (sqrtroLL + sqrtroRR);

        double wLLL = (sqrtroLL * wLL + sqrtroRR * wRR + //
            idbn * (bmm2 - bmm1)) / (sqrtroLL + sqrtroRR);

        double bttt = (sqrtroLL * btt2 + sqrtroRR * btt1 + //
            idbn * sqrtroLL * sqrtroRR * (vRR - vLL)) / (sqrtroLL + sqrtroRR);

        double bmmm = (sqrtroLL * bmm2 + sqrtroRR * bmm1 + //
            idbn * sqrtroLL * sqrtroRR * (wRR - wLL)) / (sqrtroLL + sqrtroRR);

        double eLLL = eLL - idbn * sqrtroLL * ((SM * bn + vLL * btt1 + wLL * bmm1) //
            - (SM * bn + vLLL * bttt + wLLL * bmmm));
        double eRRR = eRR + idbn * sqrtroRR * ((SM * bn + vRR * btt2 + wRR * bmm2) //
            - (SM * bn + vLLL * bttt + wLLL * bmmm));
        //cout << " = " << bn << " " << btt2 << " " << bmm2 << endl;
        //cout << "sbvr = " << (SM * bn + vRR * btt2 + wRR * bmm2) << endl;
        double  ULL[9], URR[9], ULLL[9], URRR[9];

        ULL[0] = ro_LL;
        ULL[1] = ro_LL * SM;
        ULL[2] = ro_LL * vLL;
        ULL[3] = ro_LL * wLL;
        ULL[4] = eLL;
        ULL[5] = bn;
        ULL[6] = btt1;
        ULL[7] = bmm1;
        ULL[8] = Q_LL;

        URR[0] = ro_RR;
        //cout << ro_RR << endl;
        URR[1] = ro_RR * SM;
        URR[2] = ro_RR * vRR;
        URR[3] = ro_RR * wRR;
        URR[4] = eRR;
        URR[5] = bn;
        URR[6] = btt2;
        URR[7] = bmm2;
        URR[8] = Q_RR;

        ULLL[0] = ro_LL;
        ULLL[1] = ro_LL * SM;
        ULLL[2] = ro_LL * vLLL;
        ULLL[3] = ro_LL * wLLL;
        ULLL[4] = eLLL;
        ULLL[5] = bn;
        ULLL[6] = bttt;
        ULLL[7] = bmmm;
        ULLL[8] = Q_LL;

        URRR[0] = ro_RR;
        URRR[1] = ro_RR * SM;
        URRR[2] = ro_RR * vLLL;
        URRR[3] = ro_RR * wLLL;
        URRR[4] = eRRR;
        URRR[5] = bn;
        URRR[6] = bttt;
        URRR[7] = bmmm;
        URRR[8] = Q_RR;

        double PO[9];

        if (SL >= 0.0)
        {
            //cout << "SL >= 0.0" << endl;
            for (int i = 0; i < 9; i++)
            {
                PO[i] = FL[i];
            }
        }
        else if (SL < 0.0 && SLL >= 0.0)
        {
            //cout << "SL < 0.0 && SLL >= 0.0" << endl;
            for (int i = 0; i < 9; i++)
            {
                PO[i] = FL[i] + SL * ULL[i] - SL * UL[i];
            }
            //cout << ULL[0] << endl;
        }
        else if (SLL <= 0.0 && SM >= 0.0)
        {
            //cout << "SLL <= 0.0 && SM >= 0.0" << endl;
            for (int i = 0; i < 9; i++)
            {
                PO[i] = FL[i] + SLL * ULLL[i] - (SLL - SL) * ULL[i] - SL * UL[i];
            }
        }
        else if (SM < 0.0 && SRR > 0.0)
        {
            //cout << "SM < 0.0 && SRR > 0.0" << endl;
            for (int i = 0; i < 9; i++)
            {
                PO[i] = FR[i] + SRR * URRR[i] - (SRR - SR) * URR[i] - SR * UR[i];
            }
            //cout << "P4 = " << URRR[4] << endl;
        }
        else if (SR > 0.0 && SRR <= 0.0)
        {
            //cout << "SR > 0.0 && SRR <= 0.0" << endl;
            for (int i = 0; i < 9; i++)
            {
                PO[i] = FR[i] + SR * URR[i] - SR * UR[i];
            }
            //cout << URR[0] << endl;
        }
        else if (SR <= 0.0)
        {
            //cout << "SR <= 0.0" << endl;
            for (int i = 0; i < 9; i++)
            {
                PO[i] = FR[i];
            }
        }



        double SN = max(fabs(SL), fabs(SR));

        PO[5] = -SN * (bn2 - bn1);

        P[1] = n1 * PO[1] + t1 * PO[2] + m1 * PO[3];
        P[2] = n2 * PO[1] + t2 * PO[2] + m2 * PO[3];
        P[3] = n3 * PO[1] + t3 * PO[2] + m3 * PO[3];
        P[5] = spi4 * (n1 * PO[5] + t1 * PO[6] + m1 * PO[7]);
        P[6] = spi4 * (n2 * PO[5] + t2 * PO[6] + m2 * PO[7]);
        P[7] = spi4 * (n3 * PO[5] + t3 * PO[6] + m3 * PO[7]);
        P[0] = PO[0];
        P[4] = PO[4];
        PQ = PO[8];

        double SWAP = P[4];
        P[4] = P[5];
        P[5] = P[6];
        P[6] = P[7];
        P[7] = SWAP;
        return time;
    }

}

__device__ double linear(const double& x1, const double& t1, const double& x2, const double& t2, const double& y)
{
    double d = (t1 - t2) / (x1 - x2);
    return  (d * (y - x2) + t2);
}

__device__ double minmod(const double& x, const double& y)
{
    if (sign(x) + sign(y) == 0)
    {
        return 0.0;
    }
    else
    {
        return   ((sign(x) + sign(y)) / 2.0) * min(fabs(x), fabs(y));  ///minmod
        //return (2*x*y)/(x + y);   /// vanleer
    }
}

__device__ double sign(const double& x)
{
    if (x > 0)
    {
        return 1.0;
    }
    else if (x < 0)
    {
        return -1.0;
    }
    else
    {
        return 0.0;
    }
}

__device__ double linear(const double& x1, const double& t1, const double& x2, const double& t2, const double& x3, const double& t3, const double& y)
// ������� �������� � ����������� 2
// ������ ����� ����� 1 � 2,  2 � 3, ����� ������� �������� �������� � y
{
    double d = minmod((t1 - t2) / (x1 - x2), (t2 - t3) / (x2 - x3));
    return  (d * (y - x2) + t2);
}

void takeDataOmni(double* ro, double* p, double* u, double* t, int n1, int n2)
{
    ifstream fout1;
    fout1.open("omni_all_data_day.txt"); // omni_all_data.txt   omni_all_data_hour.txt     omni_all_data_day.txt
    int a1, a2, a3, a4, a5;
    double b1, b2, b3, b4;
    string s1, s2, s3, s4;
    fout1 >> s1 >> s2 >> s3 >> s4;
    cout << s1 << endl;
    for (int i = 0; i < n2; i++)
    {
        fout1 >> a1 >> b1 >> b2 >> b3;
    }

    for (int i = 0; i < n1; i++)
    {
        fout1 >> a1 >> b1 >> b2 >> b3;
        t[i] = a1 * minut;
        u[i] = b1 / 375.0;
        ro[i] = b2 / 7.0;
        p[i] = 2.0 * ro[i] * b3 * 5.86922 * 0.00000001;
    }
    fout1.close();
}

__global__ void takeOmni(double* ro, double* p, double* ro_p, double* p_p, double* u, double* RO, double* P, double* U, double* T, double* t_now, int* dev_mas_Omni)
{
    //printf("Omni 2,   %lf,   %lf,   %d \n", T[0], *t_now, *dev_mas_Omni);
    for (int k = *dev_mas_Omni; k < Omni_; k++)
    {
        if (T[k] >= *t_now && k >= 1)
        {
            *dev_mas_Omni = k;
            ro[0] = linear(T[k - 1], RO[k - 1], T[k], RO[k], *t_now);
            u[0] = linear(T[k - 1], U[k - 1], T[k], U[k], *t_now);
            p[0] = linear(T[k - 1], P[k - 1], T[k], P[k], *t_now);
            ro_p[0] = ro[0]; // 0.000001;
            p_p[0] = p[0];
            //printf("Omni,   %lf,   %lf,   %lf,  %d \n", T[k], *t_now, T[k - 1], *dev_mas_Omni);
            break;
        }
    }

}

__global__ void takeVoyadger(double* T_do, double* t_now, double* T_V, int* dev_mas_V, int* voy)
{
    for (int k = *dev_mas_V; k < 127476; k++)
    {
        if (T_V[k] > *t_now)
        {
            *dev_mas_V = k;
            break;
        }
    }

    if (*T_do > T_V[*dev_mas_V] - *t_now)
    {
        *T_do = T_V[*dev_mas_V] - *t_now;
        *voy = 1;
    }
    else
    {
        *voy = 0;
    }

}

__global__ void takeVoyadger2(double* ro, double* ro2, double* p, double* u, double* distV, int* dev_mas_V, double* j1, double* j2, double* j3)
{
    int kk = int((distV[*dev_mas_V] - 1.0) / dx);
    double r1 = L + kk * dx;
    double r2 = L + (kk + 1) * dx;
    //*j1 = linear(r1, ro[kk] - ro2[kk], r2, ro[kk + 1] - ro2[kk + 1], distV[*dev_mas_V]);
    *j1 = linear(r1, ro2[kk], r2, ro2[kk + 1], distV[*dev_mas_V]);
    *j2 = linear(r1, u[kk], r2, u[kk + 1], distV[*dev_mas_V]);
    *j3 = linear(r1, p[kk], r2, p[kk + 1], distV[*dev_mas_V]);
}

__global__ void init_time(double* T_do, double* T)
{
    *T_do = 0.001 * minut;
    *T = 0.001 * minut;
}

__global__ void calcul(double* ro, double* u, double* p, double* ro2, double* u2, double* p2,//
    double* T, double* T_do, int* N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;   // ���������� ������ ������� ������ (�������� ������)
    
    if (i >= *N)
    {
        return;
    }

    if (i == 0) // Ƹ����� ��������� �������
    {
        return;
    }

    if (i == *N - 1) // Ƹ����� ��������� �������
    {
        ro2[*N - 1] = ro[*N - 2];
        p2[*N - 1] = p[*N - 2];
        u2[*N - 1] = u[*N - 2];
        return;
    }

    double P[5] = { 0.0 };
    double r = L + i * dx;
    double B1 = 0.0;
    double B2 = 0.0;
    double B3 = 0.0;
    double time2 = 0.1 * minut;
    double ro1 = ro[i];
    double p1 = p[i];
    double u1 = u[i];
    double ro3 = ro[i + 1];
    double p3 = p[i + 1];
    double u3 = u[i + 1];
    double ro4 = ro[i - 1];
    double p4 = p[i - 1];
    double u4 = u[i - 1];

    double roL = ro1;
    double pL = p1;
    double uL = u1;
    double roR = ro3;
    double pR = p3;
    double uR = u3;
    double PQ;

    if (i > 1 && i < *N - 2)
    {
        roL = linear(r - dx, ro4, r, ro1, r + dx, ro3, r + dx / 2.0);
        pL = linear(r - dx, p4, r, p1, r + dx, p3, r + dx / 2.0);
        uL = linear(r - dx, u4, r, u1, r + dx, u3, r + dx / 2.0);
        if (roL <= 0.0)
        {
            roL = ro1;
        }
        if (pL <= 0.0)
        {
            pL = p1;
        }

        roR = linear(r, ro1, r + dx, ro3, r + 2.0 * dx, ro[i + 2], r + dx / 2.0);
        pR = linear(r, p1, r + dx, p3, r + 2.0 * dx, p[i + 2], r + dx / 2.0);
        uR = linear(r, u1, r + dx, u3, r + 2.0 * dx, u[i + 2], r + dx / 2.0);
        if (roR <= 0.0)
        {
            roR = ro3;
        }
        if (pR <= 0.0)
        {
            pR = p3;
        }

    }

    double CC, RO;
    time2 = min(time2, HLLC_2d_Korolkov_b_s(roL, 1.0, pL, uL, 0.0, 1.0, roR, 1.0, pR, uR, 0.0, 1.0, 0.0, P, PQ, 1.0, 0.0, dx, CC, RO));
    B1 = P[0];
    B2 = P[1];
    B3 = P[3];  // 4

    roL = ro1;
    pL = p1;
    uL = u1;
    roR = ro4;
    pR = p4;
    uR = u4;

    if (i > 1 && i < *N - 2)
    {
        roL = linear(r - dx, ro4, r, ro1, r + dx, ro3, r - dx / 2.0);
        pL = linear(r - dx, p4, r, p1, r + dx, p3, r - dx / 2.0);
        uL = linear(r - dx, u4, r, u1, r + dx, u3, r - dx / 2.0);
        if (roL <= 0.0)
        {
            roL = ro1;
        }
        if (pL <= 0.0)
        {
            pL = p1;
        }

        roR = linear(r - 2.0 * dx, ro[i - 2], r - dx, ro4, r, ro1, r - dx / 2.0);
        pR = linear(r - 2.0 * dx, p[i - 2], r - dx, p4, r, p1, r - dx / 2.0);
        uR = linear(r - 2.0 * dx, u[i - 2], r - dx, u4, r, u1, r - dx / 2.0);
        if (roR <= 0.0)
        {
            roR = ro4;
        }
        if (pR <= 0.0)
        {
            pR = p4;
        }

    }


    time2 = min(time2, HLLC_2d_Korolkov_b_s(roL, 1.0, pL, uL, 0.0, 1.0, roR, 1.0, pR, uR, 0.0, 1.0, 0.0, P, PQ, -1.0, 0.0, dx, CC, RO));

    B1 = B1 + P[0];
    B2 = B2 + P[1];
    B3 = B3 + P[3];  // 4

    ro2[i] = -*T_do * (B1 / dx - (2.0 / r) * (-ro1 * u1)) + ro1;
    if (ro2[i] <= 0.0)
    {
        printf("Error ro \n");
    }
    u2[i] = (-*T_do * (B2 / dx + Kn * n_H * exp(-r0/r) * kv(u1) * ro1 * sigma(u1) - (2.0 / r) * (-ro1 * u1 * u1)) + ro1 * u1) / ro2[i];
    //p2[i] = ((-*T_do * (B3 / dx + 0.5 * Kn * n_H * exp(-r0 / r) * kv(u1) * u1 * ro1 * sigma(u1) - (2.0 / r) * (-(ggg * p1 * u1) / (ggg - 1.0) - ro1 * u1 * u1 * u1 / 2.0)) + //
    //    p1 / (ggg - 1.0) + ro1 * u1 * u1 / 2.0) - ro2[i] * u2[i] * u2[i] / 2.0) * (ggg - 1.0);
    //u2[i] = (-*T_do * (B2 / dx - (2.0 / r) * (-ro1 * u1 * u1)) + ro1 * u1) / ro2[i];
    //p2[i] = ((-*T_do * (B3 / dx - (2.0 / r) * (-(ggg * p1 * u1) / (ggg - 1.0) - ro1 * u1 * u1 * u1 / 2.0)) + //
    //    p1 / (ggg - 1.0) + ro1 * u1 * u1 / 2.0) - ro2[i] * u2[i] * u2[i] / 2.0) * (ggg - 1.0);
    p2[i] = ((-*T_do * (B3 / dx + Kn * n_H * exp(-r0 / r) * kv(u1) * u1 * ro1 * sigma(u1) - (2.0 / r) * (-(ggg * p1 * u1) / (ggg - 1.0) - ro1 * u1 * u1 * u1 / 2.0)) + //
        p1 / (ggg - 1.0) + ro1 * u1 * u1 / 2.0) - ro2[i] * u2[i] * u2[i] / 2.0) * (ggg - 1.0);

    if (p2[i] <= 0.0)
    {
        p2[i] = 0.000001;
    }

    if (time2 < *T)
    {
        *T = time2;
    }

}

__global__ void calcul_component(double* ro, double* u, double* p, double* ro2, double* u2, double* p2, double* ro_p, double* ro_p2, double* p_p, double* p_p2,//
    double* T, double* T_do, int* N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;   // ���������� ������ ������� ������ (�������� ������)

    if (i >= *N)
    {
        return;
    }

    if (i == 0) // Ƹ����� ��������� �������
    {
        ro2[0] = ro[0];
        p2[0] = p[0];
        u2[0] = u[0];
        ro_p2[0] = ro_p[0];
        p_p2[0] = p_p[0];
        return;
    }

    if (i == *N - 1) // ������ ��������� �������
    {
        ro2[*N - 1] = ro[*N - 2];
        p2[*N - 1] = p[*N - 2];
        u2[*N - 1] = u[*N - 2];

        ro_p2[*N - 1] = ro_p[*N - 2];
        p_p2[*N - 1] = p_p[*N - 2];
        return;
    }

    double P[5] = { 0.0 };
    double r = L + i * dx;
    double B1 = 0.0;
    double B2 = 0.0;
    double B3 = 0.0;
    double Bp1 = 0.0;
    double Bp2 = 0.0;
    double Bp3 = 0.0;
    double B_pi1 = 0.0;
    double B_p3 = 0.0;
    double time2 = 0.3 * minut;  // 0.3
    double ro1 = ro[i];
    double p1 = p[i];
    double ro_p1 = ro_p[i];
    double p_p1 = p_p[i];
    double u1 = u[i];
    double ro3 = ro[i + 1];
    double p3 = p[i + 1];
    double ro_p3 = ro_p[i + 1];
    double p_p3 = p_p[i + 1];
    double u3 = u[i + 1];
    double ro4 = ro[i - 1];
    double p4 = p[i - 1];
    double ro_p4 = ro_p[i - 1];
    double p_p4 = p_p[i - 1];
    double u4 = u[i - 1];

    double roL = ro1;
    double pL = p1;
    double ro_pL = ro_p1;
    double p_pL = p_p1;
    double uL = u1;
    double roR = ro3;
    double pR = p3;
    double ro_pR = ro_p3;
    double p_pR = p_p3;
    double uR = u3;
    double PQ;

    if (i > 1 && i < *N - 2)
    {
        roL =   linear(r - dx, ro4,   r, ro1,   r + dx, ro3,   r + dx / 2.0);
        pL =    linear(r - dx, p4,    r, p1,    r + dx, p3,    r + dx / 2.0);
        ro_pL = linear(r - dx, ro_p4, r, ro_p1, r + dx, ro_p3, r + dx / 2.0);
        p_pL =  linear(r - dx, p_p4,  r, p_p1,  r + dx, p_p3,  r + dx / 2.0);
        uL =    linear(r - dx, u4,    r, u1,    r + dx, u3,    r + dx / 2.0);
        if (roL <= 0.0)
        {
            roL = ro1;
        }
        if (pL <= 0.0)
        {
            pL = p1;
        }
        if (ro_pL <= 0.0)
        {
            ro_pL = ro_p1;
        }
        if (p_pL <= 0.0)
        {
            p_pL = p_p1;
        }

        roR =   linear(r, ro1,   r + dx, ro3,   r + 2.0 * dx, ro[i + 2],    r + dx / 2.0);
        pR =    linear(r, p1,    r + dx, p3,    r + 2.0 * dx, p[i + 2],     r + dx / 2.0);
        ro_pR = linear(r, ro_p1, r + dx, ro_p3, r + 2.0 * dx, ro_p[i + 2],  r + dx / 2.0);
        p_pR =  linear(r, p_p1,  r + dx, p_p3,  r + 2.0 * dx, p_p[i + 2],   r + dx / 2.0);
        uR =    linear(r, u1,    r + dx, u3,    r + 2.0 * dx, u[i + 2],     r + dx / 2.0);
        if (roR <= 0.0)
        {
            roR = ro3;
        }
        if (pR <= 0.0)
        {
            pR = p3;
        }
        if (ro_pR <= 0.0)
        {
            ro_pR = ro_p3;
        }
        if (p_pR <= 0.0)
        {
            p_pR = p_p3;
        }

    }

    double PP, RO; 
    time2 = min(time2, HLLC_2d_Korolkov_b_s(roL, ro_pL, pL, uL, 0.0, p_pL, roR, ro_pR, pR, uR, 0.0, p_pR, 0.0, P, PQ, 1.0, 0.0, dx, RO, PP));
    B1 = P[0];
    B2 = P[1];
    B3 = P[3];  // 4

    time2 = min(time2, HLLC_2d_Korolkov_b_s(ro_pL, 1.0, p_pL, uL, 0.0, 1.0, ro_pR, 1.0, p_pR, uR, 0.0, 1.0, 0.0, P, PQ, 1.0, 0.0, dx, RO, PP));
    Bp1 = P[0];
    Bp2 = P[1];
    Bp3 = P[3];  // 4

    B_pi1 = PQ;
    B_p3 = PP/pow(RO, ggg);// P[4];

    roL = ro1;
    pL = p1;
    ro_pL = ro_p1;
    p_pL = p_p1;
    uL = u1;
    roR = ro4;
    pR = p4;
    ro_pR = ro_p4;
    p_pR = p_p4;
    uR = u4;

    if (i > 1 && i < *N - 2)
    {
        roL =   linear(r - dx, ro4,   r, ro1,   r + dx, ro3,   r - dx / 2.0);
        pL =    linear(r - dx, p4,    r, p1,    r + dx, p3,    r - dx / 2.0);
        ro_pL = linear(r - dx, ro_p4, r, ro_p1, r + dx, ro_p3, r - dx / 2.0);
        p_pL =  linear(r - dx, p_p4,  r, p_p1,  r + dx, p_p3,  r - dx / 2.0);
        uL =    linear(r - dx, u4,    r, u1,    r + dx, u3,    r - dx / 2.0);
        if (roL <= 0.0)
        {
            roL = ro1;
        }
        if (pL <= 0.0)
        {
            pL = p1;
        }
        if (ro_pL <= 0.0)
        {
            ro_pL = ro_p1;
        }
        if (p_pL <= 0.0)
        {
            p_pL = p_p1;
        }

        roR =   linear(r - 2.0 * dx, ro[i - 2],   r - dx, ro4,   r, ro1,   r - dx / 2.0);
        pR =    linear(r - 2.0 * dx, p[i - 2],    r - dx, p4,    r, p1,    r - dx / 2.0);
        ro_pR = linear(r - 2.0 * dx, ro_p[i - 2], r - dx, ro_p4, r, ro_p1, r - dx / 2.0);
        p_pR =  linear(r - 2.0 * dx, p_p[i - 2],  r - dx, p_p4,  r, p_p1,  r - dx / 2.0);
        uR =    linear(r - 2.0 * dx, u[i - 2],    r - dx, u4,    r, u1,    r - dx / 2.0);
        if (roR <= 0.0)
        {
            roR = ro4;
        }
        if (pR <= 0.0)
        {
            pR = p4;
        }
        if (ro_pR <= 0.0)
        {
            ro_pR = ro_p4;
        }
        if (p_pR <= 0.0)
        {
            p_pR = p_p4;
        }

    }

    PP = 0.0;
    RO = 0.0;
    time2 = min(time2, HLLC_2d_Korolkov_b_s(roL, ro_pL, pL, uL, 0.0, p_pL, roR, ro_pR, pR, uR, 0.0, p_pR, 0.0, P, PQ, -1.0, 0.0, dx, RO, PP));

    B1 = B1 + P[0];
    B2 = B2 + P[1];
    B3 = B3 + P[3];  // 4

    time2 = min(time2, HLLC_2d_Korolkov_b_s(ro_pL, 1.0, p_pL, uL, 0.0, 1.0, ro_pR, 1.0, p_pR, uR, 0.0, 1.0, 0.0, P, PQ, -1.0, 0.0, dx, RO, PP));

    Bp1 = Bp1 + P[0];
    Bp2 = Bp2 + P[1];
    Bp3 = Bp3 + P[3];

    B_pi1 = B_pi1 + PQ;

    B_p3 = B_p3 - PP / pow(RO, ggg);// P[4];

    //B_pi1 = (ro_p1 * u1 - ro_p4 * u4);

    //ro2[i] = -*T_do * (B1 / dx - (2.0 / r) * (-ro1 * u1)) + ro1;
    //if (ro2[i] <= 0.0)
    //{
    //    printf("Error ro \n");
    //}
    //u2[i] = (-*T_do * (B2 / dx + Kn * n_H * exp(-r0 / r) * kv(u1) * ro1 * sigma(u1) - (2.0 / r) * (-ro1 * u1 * u1)) + ro1 * u1) / ro2[i];
    //u2[i] = (-*T_do * (B2 / dx - (2.0 / r) * (-ro1 * u1 * u1)) + ro1 * u1) / ro2[i];
    //p2[i] = ((-*T_do * (B3 / dx - (2.0 / r) * (-(ggg * p1 * u1) / (ggg - 1.0) - ro1 * u1 * u1 * u1 / 2.0)) + //
    //    p1 / (ggg - 1.0) + ro1 * u1 * u1 / 2.0) - ro2[i] * u2[i] * u2[i] / 2.0) * (ggg - 1.0);
    
    //p2[i] = ((-*T_do * (B3 / dx + Kn * n_H * exp(-r0 / r) * kv(u1) * u1 * ro1 * sigma(u1) - (2.0 / r) * (-(ggg * p1 * u1) / (ggg - 1.0) - ro1 * u1 * u1 * u1 / 2.0)) + //
    //    p1 / (ggg - 1.0) + ro1 * u1 * u1 / 2.0) - ro2[i] * u2[i] * u2[i] / 2.0) * (ggg - 1.0);


    // ������� ��������� 2-������

    double pp = p_p1;
    double roo = ro_p1;

    double roH = n_H * exp(-r0 / r);
    double UH = sqrt(kv(v_H - u1) + 4.0 / pi * (kv(c_H) + pp / roo));
    double UMH = sqrt(kv(v_H - u1) + 64.0 / (9.0 * pi) * (kv(c_H) + pp / roo));
    double nu = Kn * roo * roH * UMH * sigma(UMH);
    double Q1, Q2, Q3, Q1pi, Q3p;
    Q1 = 0.0;
    Q2 = nu* (v_H - u1);
    Q3 = nu* (0.5 * (kv(v_H) - kv(u1)) + (kv(c_H) - pp / roo) * UH / UMH);
    Q1pi = -Kn * roo * roH * UH * sigma(UH);// +0.0 * nu * (0.5 * (kv(v_H - u1)) + (kv(c_H) - p_pp / ro_pp) * UH / UMH);
    Q3p = 0.02 * nu * (0.5 * (kv(v_H - u1)) + (kv(c_H) - pp / roo) * UH / UMH);;

    //printf("%lf, %lf\n", Q3, Q3p);

    ro2[i] = -*T_do * (B1 / dx - Q1 - (2.0 / r) * (-ro1 * u1)) + ro1;
    u2[i] = (-*T_do * (B2 / dx - Q2 - (2.0 / r) * (-ro1 * u1 * u1)) + ro1 * u1) / ro2[i];
    p2[i] = ((-*T_do * (B3 / dx - Q3 - (2.0 / r) * (-(ggg * p1 * u1) / (ggg - 1.0) - ro1 * u1 * u1 * u1 / 2.0)) + //
        p1 / (ggg - 1.0) + ro1 * u1 * u1 / 2.0) - ro2[i] * u2[i] * u2[i] / 2.0) * (ggg - 1.0);


    ro_p2[i] = -*T_do * (Bp1 / dx - Q1pi - (2.0 / r) * (-ro_p1 * u1)) + ro_p1;
    double Q = (ro_p2[i] * u2[i] - ro_p1 * u1) / *T_do + Bp2 / dx - (2.0 / r) * (-ro_p1 * u1 * u1);
    p_p2[i] = ((-*T_do * (Bp3 / dx - (Q3p - Q1pi * (0.5 * kv(u1) + p_p1 / g1) + Q * u1) - (2.0 / r) * (-(ggg * p_p1 * u1) / (ggg - 1.0) - ro_p1 * u1 * u1 * u1 / 2.0)) + //
        p_p1 / (ggg - 1.0) + ro_p1 * u1 * u1 / 2.0) - ro_p2[i] * u2[i] * u2[i] / 2.0) * (ggg - 1.0);
    

    //ro_p2[i] = ro_p1; // -*T_do * (B_pi1 / dx - Q1pi - (2.0 / r) * (-ro_p1 * u1)) + ro_p1; // ����� ��������� �����-�����

    //u2[i] = (-*T_do * (B_p2 / dx - Q2 - (2.0 / r) * (-ro_p1 * u1 * u1)) + ro_p1 * u1) / ro_p2[i];
    
    //p_p2[i] = ((-*T_do * (B_p3 / dx - Q3p - (2.0 / r) * (-(ggg * p_p1 * u1) / (ggg - 1.0) - (ro1 - ro_p1) *u1 * u1 * u1 / 2.0)) + //
    //    p_p1 / (ggg - 1.0) + (ro1 - ro_p1) * u1 * u1 / 2.0) - (ro2[i] - ro_p2[i]) * u2[i] * u2[i] / 2.0) * (ggg - 1.0);

    //p_p2[i] = *T_do * (ggg - 1.0) *  (-(1.0 / (ggg - 1.0)) * (p_p1 * u1 - p_p4 * u4) / dx - (ggg / (ggg - 1.0)) * (2.0 * p_p1 * u1 / r) - p_p1 * (u1 - u4) / dx) + p_p1;

    //p_p2[i] = p_p1;// (-*T_do * u1 * (B_p3) / dx + p_p1 / pow(ro1 - ro_p1, ggg))* pow(ro2[i] - ro_p2[i], ggg);


    if (p_p2[i] < 0.0)
    {
        printf("ERROR  707  p < 0 = %lf __  %lf \n", r, p_p2[i]);
        p_p2[i] = p2[i];
    }

    if (ro_p2[i] < 0.0)
    {
        printf("ERROR  707  ro < 0 = %lf __  %lf \n", r, ro_p2[i]);
        ro_p2[i] = ro2[i];
    }

    if (time2 < *T)
    {
        *T = time2;
    }

}


int main()
{
    double* ro, * p, * u;

    double* swap_;
    int N = 8800;



    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(ro, p, u, N);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }


    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(double *ro, double* p, double* u, int& N)
{
    cout << "Start -2" << endl;
    double* ro2, * p2;
    double* dev_ro, * dev_p, * dev_u, * dev_ro2, * dev_p2, * dev_u2, * dev_ro_p, * dev_ro_p2, * dev_p_p, * dev_p_p2;
    double* ro_, * p_, * u_, * rop_, * pp_;
    double* dev_T_all;
    double* dev_T_do;
    double* T_all;
    double* T_do, * T, * dev_T;
    double* Time_Omni, * Ro_Omni, * P_Omni, * U_Omni;
    double* Time_V, * Ro_V, * P_V, * U_V, * Dist_V;
    double* dev_Time_Omni, * dev_Ro_Omni, * dev_P_Omni, * dev_U_Omni;
    double* dev_Time_V, * dev_Ro_V, * dev_P_V, * dev_U_V, * dev_Dist_V;
    hipError_t cudaStatus;
    int N_V = 127476;               // ����� ������ ��������, ������ ���� �������������
    string s1, s2, s3, s4, s5, s6, s7, s8;
    int a1, a2, a3, a4;
    double b1, b2, b3, b4;
    int N_O1 = Omni_, N_O2 = 0;     // ������ ����-������� ������ ���� ���������� (������ ��� �� ���� ��� �������� �� 100.000)
    int* dev_mas_V, * mas_V;
    int* dev_mas_Omni, * mas_Omni;
    int* dev_N;
    int* NN;
    double* j1, * j2, * j3, * dev_j1, * dev_j2, * dev_j3;
    ofstream fout2;
    int step = 0;
    int* voy, * dev_voy;  // �������� �� ������ ��������?
    ofstream fout5;
    fout5.open("voyadger2_calculations.txt");// , ios_base::out | ios_base::app);

    cout << "Start -1" << endl;

    NN = new int[1];
    *NN = N;
    Time_V = new double[N_V];
    Dist_V = new double[N_V];
    U_V = new double[N_V];
    Ro_V = new double[N_V];
    P_V = new double[N_V];
    T_all = new double[1];
    Ro_Omni = new double[N_O1];
    P_Omni = new double[N_O1];
    U_Omni = new double[N_O1];
    Time_Omni = new double[N_O1];
    mas_Omni = new int[1];
    ro = new double[N];
    p = new double[N];
    u = new double[N];
    ro2 = new double[N];
    p2 = new double[N];
    ro_ = new double[N];
    p_ = new double[N];
    rop_ = new double[N];
    pp_ = new double[N];
    u_ = new double[N];
    voy = new int[1];
    mas_V = new int[1];
    j1 = new double[1];
    j2 = new double[1];
    j3 = new double[1];

    *voy = 0;
    *mas_V = 0;

    *T_all = 50.0; //  50.0; // 3600.01 * minut;

    cout << "Start 0" << endl;

    takeDataOmni(Ro_Omni, P_Omni, U_Omni, Time_Omni, N_O1, N_O2);

    cout << "Start 01" << endl;
    // ��������� ��������� �������

    for (int i = 0; i < N; i++)
    {
        double r = L + i * dx;
        double pE = 0.0059994;
        ro[i] = 1.0 / (r * r);
        ro2[i] = 1.0 / (r * r); //0.000001 / (r * r);
        p[i] =  pE * pow(1.0 / r, 2.0 * ggg);
        u[i] = 1.0;
    }

    cout << "Start 1" << endl;

    // ��������� ������ ��������
    if (true)
    {
        ifstream fout4;
        fout4.open("voyager2_all_data.txt");
        fout4 >> s1 >> s2 >> s3 >> s4 >> s5 >> s6 >> s7 >> s8;
        for (int i = 0; i < N_V; i++)
        {
            fout4 >> a1 >> a2 >> a3 >> a4;
            fout4 >> b1 >> b2 >> b3 >> b4;
            Time_V[i] = a1 * hour;
            Dist_V[i] = b1;
            U_V[i] = b2;
            Ro_V[i] = b3;
            P_V[i] = b4;
        }
        fout4.close();
    }

    cout << "Start 2" << endl;

    // �������� ������ �� CUDA
    if (true)
    {
        // Choose which GPU to run on, change this on a multi-GPU system.
        cudaStatus = hipSetDevice(0);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
            goto Error;
        }

        // Allocate GPU buffers for three vectors (two input, one output)    .
        cudaStatus = hipMalloc((void**)&dev_ro, N * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_p, N * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_u, N * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_ro2, N * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_p2, N * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_u2, N * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_Time_V, N_V * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_Dist_V, N_V * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_Time_Omni, N_O1 * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_Ro_Omni, N_O1 * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_P_Omni, N_O1 * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_U_Omni, N_O1 * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_T_all, sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_T, sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_T_do, sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_mas_V, sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_N, sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_mas_Omni, sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_mas_V, sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_voy, sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_j1, sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_j2, sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_j3, sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_ro_p, N * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_p_p, N * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_ro_p2, N * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }

        cudaStatus = hipMalloc((void**)&dev_p_p2, N * sizeof(double));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }
    }

    cout << "Start 3" << endl;

    // �������� ������ �� CUDA
    if (true)
    {
        cudaStatus = hipMemcpy(dev_Time_V, Time_V, N_V * sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_Dist_V, Dist_V, N_V * sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_Time_Omni, Time_Omni, N_O1 * sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_U_Omni, U_Omni, N_O1 * sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_Ro_Omni, Ro_Omni, N_O1 * sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_P_Omni, P_Omni, N_O1 * sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_ro, ro, N * sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_ro2, ro, N * sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_p, p, N * sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_p2, p, N * sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_u, u, N * sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_u2, u, N * sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_T_all, T_all, sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_N, NN, sizeof(int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_mas_V, mas_V, sizeof(int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_voy, voy, sizeof(int), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_ro_p, ro2, N * sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_ro_p2, ro2, N * sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_p_p, p, N * sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }

        cudaStatus = hipMemcpy(dev_p_p2, p, N * sizeof(double), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }
    }

    cout << "Start 4" << endl;

    init_time << <1, 1 >> > (dev_T_do, dev_T);   // ������������� ������ ��� �� �������

    while (*mas_V < 127470) // (*T_all < 60.0)//
    {
        step++;
        takeOmni << <1, 1 >> > (dev_ro, dev_p, dev_ro_p, dev_p_p, dev_u, dev_Ro_Omni, dev_P_Omni, dev_U_Omni, dev_Time_Omni, dev_T_all, dev_mas_Omni);
       
        takeVoyadger << <1, 1 >> > (dev_T_do, dev_T_all, dev_Time_V, dev_mas_V, dev_voy);

        // Launch a kernel on the GPU with one thread for each element.
        //calcul << < (int)(1.0 * *NN / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (dev_ro, dev_u, dev_p, dev_ro2, dev_u2, dev_p2,//
        //    dev_T, dev_T_do, dev_N);

        calcul_component << < (int)(1.0 * *NN / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (dev_ro, dev_u, dev_p, dev_ro2, dev_u2, dev_p2,//
            dev_ro_p, dev_ro_p2, dev_p_p, dev_p_p2, dev_T, dev_T_do, dev_N);

        funk_time << <1, 1 >> > (dev_T, dev_T_do, dev_T_all);

        cudaStatus = hipMemcpy(voy, dev_voy, sizeof(int), hipMemcpyDeviceToHost);
        
        // ����� ���������� ������ ��������
        if (*voy == 1)
        {
            *voy == 0;
            cudaStatus = hipMemcpy(dev_voy, voy, sizeof(int), hipMemcpyHostToDevice);
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "hipMemcpy failed 5!");
                goto Error;
            }

            cudaStatus = hipMemcpy(mas_V, dev_mas_V, sizeof(int), hipMemcpyDeviceToHost);
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "hipMemcpy failed 6!");
                goto Error;
            }

            takeVoyadger2 << <1, 1 >> > (dev_ro2, dev_ro_p2, dev_p_p2, dev_u2, dev_Dist_V, dev_mas_V, dev_j1, dev_j2, dev_j3);
            cudaStatus = hipMemcpy(j1, dev_j1, sizeof(double), hipMemcpyDeviceToHost);
            cudaStatus = hipMemcpy(j2, dev_j2, sizeof(double), hipMemcpyDeviceToHost);
            cudaStatus = hipMemcpy(j3, dev_j3, sizeof(double), hipMemcpyDeviceToHost);
            cudaStatus = hipMemcpy(T_all, dev_T_all, sizeof(double), hipMemcpyDeviceToHost);
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "hipMemcpy failed!");
                goto Error;
            }
            fout5 << *T_all << " " << Dist_V[*mas_V] << " " << *j1 * 7.0 << " " << *j2 * 375.0 << " " << *j3 / (2.0 * *j1 * 5.86922 * 0.00000001) << //
                " " << Ro_V[*mas_V] << " " << U_V[*mas_V] << " " << P_V[*mas_V] << endl;

            // � ���� ������ ����� �������� ������ ������ ��� �������� (��������� NN)
            // ��������� � ���� ������
            if (Dist_V[*mas_V] > * NN * dx - 8.0)
            {
                int kl = (int)((Dist_V[*mas_V] + 16.0) / dx);
                cout << "New size do " << kl * dx <<  "  " << kl << " " << *NN << endl;
               
                cudaStatus = hipMemcpy(ro_, dev_ro2, *NN * sizeof(double), hipMemcpyDeviceToHost);
                if (cudaStatus != hipSuccess) {
                    fprintf(stderr, "hipMemcpy failed!");
                    goto Error;
                }
               
                cudaStatus = hipMemcpy(p_, dev_p2, *NN * sizeof(double), hipMemcpyDeviceToHost);
                if (cudaStatus != hipSuccess) {
                    fprintf(stderr, "hipMemcpy failed!");
                    goto Error;
                }

                cudaStatus = hipMemcpy(rop_, dev_ro_p2, *NN * sizeof(double), hipMemcpyDeviceToHost);
                if (cudaStatus != hipSuccess) {
                    fprintf(stderr, "hipMemcpy failed!");
                    goto Error;
                }
                
                cudaStatus = hipMemcpy(pp_, dev_p_p2, *NN * sizeof(double), hipMemcpyDeviceToHost);
                if (cudaStatus != hipSuccess) {
                    fprintf(stderr, "hipMemcpy failed!");
                    goto Error;
                }
               
                cudaStatus = hipMemcpy(u_, dev_u2, *NN * sizeof(double), hipMemcpyDeviceToHost);
                if (cudaStatus != hipSuccess) {
                    fprintf(stderr, "hipMemcpy failed!");
                    goto Error;
                }
                hipFree(dev_ro);
                hipFree(dev_p);
                hipFree(dev_u);
                hipFree(dev_ro2);
                hipFree(dev_p2);
                hipFree(dev_u2);
                hipFree(dev_ro_p);
                hipFree(dev_p_p);
                hipFree(dev_ro_p2);
                hipFree(dev_p_p2);
                delete[] ro;
                delete[] p;
                delete[] ro2;
                delete[] p2;
                delete[] u;
                ro = new double[kl];
                p = new double[kl];
                ro2 = new double[kl];
                p2 = new double[kl];
                u = new double[kl];
                cudaStatus = hipMalloc((void**)&dev_ro, kl * sizeof(double));
                if (cudaStatus != hipSuccess) {
                    fprintf(stderr, "hipMalloc failed!");
                    goto Error;
                }
                cudaStatus = hipMalloc((void**)&dev_p, kl * sizeof(double));
                if (cudaStatus != hipSuccess) {
                    fprintf(stderr, "hipMalloc failed!");
                    goto Error;
                }
                cudaStatus = hipMalloc((void**)&dev_u, kl * sizeof(double));
                if (cudaStatus != hipSuccess) {
                    fprintf(stderr, "hipMalloc failed!");
                    goto Error;
                }
                cudaStatus = hipMalloc((void**)&dev_ro2, kl * sizeof(double));
                if (cudaStatus != hipSuccess) {
                    fprintf(stderr, "hipMalloc failed!");
                    goto Error;
                }
                cudaStatus = hipMalloc((void**)&dev_p2, kl * sizeof(double));
                if (cudaStatus != hipSuccess) {
                    fprintf(stderr, "hipMalloc failed!");
                    goto Error;
                }
                cudaStatus = hipMalloc((void**)&dev_u2, kl * sizeof(double));
                if (cudaStatus != hipSuccess) {
                    fprintf(stderr, "hipMalloc failed!");
                    goto Error;
                }

                cudaStatus = hipMalloc((void**)&dev_ro_p, kl * sizeof(double));
                if (cudaStatus != hipSuccess) {
                    fprintf(stderr, "hipMalloc failed!");
                    goto Error;
                }
                cudaStatus = hipMalloc((void**)&dev_p_p, kl * sizeof(double));
                if (cudaStatus != hipSuccess) {
                    fprintf(stderr, "hipMalloc failed!");
                    goto Error;
                }
                cudaStatus = hipMalloc((void**)&dev_ro_p2, kl * sizeof(double));
                if (cudaStatus != hipSuccess) {
                    fprintf(stderr, "hipMalloc failed!");
                    goto Error;
                }
                cudaStatus = hipMalloc((void**)&dev_p_p2, kl * sizeof(double));
                if (cudaStatus != hipSuccess) {
                    fprintf(stderr, "hipMalloc failed!");
                    goto Error;
                }

                for (int i = 0; i < *NN; i++)
                {
                    ro[i] = ro_[i];
                    p[i] = p_[i];
                    ro2[i] = rop_[i];
                    p2[i] = pp_[i];
                    u[i] = u_[i];
                }
                for (int i = *NN; i < kl; i++)
                {
                    double r = L + i * dx;
                    double pE = 0.0059994;
                    ro[i] = 1.0 / (r * r);
                    ro2[i] = 1.0 / (r * r);  //0.000001 / (r * r);
                    p[i] = pE * pow(1.0 / r, 2.0 * ggg);
                    p2[i] = pE * pow(1.0 / r, 2.0 * ggg);
                    u[i] = 1.0;
                }
                cudaStatus = hipMemcpy(dev_ro2, ro, kl * sizeof(double), hipMemcpyHostToDevice);
                if (cudaStatus != hipSuccess) {
                    fprintf(stderr, "hipMemcpy failed!");
                    goto Error;
                }
                cudaStatus = hipMemcpy(dev_p2, p, kl * sizeof(double), hipMemcpyHostToDevice);
                if (cudaStatus != hipSuccess) {
                    fprintf(stderr, "hipMemcpy failed!");
                    goto Error;
                }
                cudaStatus = hipMemcpy(dev_u2, u, kl * sizeof(double), hipMemcpyHostToDevice);
                if (cudaStatus != hipSuccess) {
                    fprintf(stderr, "hipMemcpy failed!");
                    goto Error;
                }
                cudaStatus = hipMemcpy(dev_ro_p2, ro2, kl * sizeof(double), hipMemcpyHostToDevice);
                if (cudaStatus != hipSuccess) {
                    fprintf(stderr, "hipMemcpy failed!");
                    goto Error;
                }
                cudaStatus = hipMemcpy(dev_p_p2, p2, kl * sizeof(double), hipMemcpyHostToDevice);
                if (cudaStatus != hipSuccess) {
                    fprintf(stderr, "hipMemcpy failed!");
                    goto Error;
                }
                N = kl;
                *NN = kl;
                cudaStatus = hipMemcpy(dev_N, NN, sizeof(int), hipMemcpyHostToDevice);
                if (cudaStatus != hipSuccess) {
                    fprintf(stderr, "hipMemcpy failed!");
                    goto Error;
                }
                delete[] ro_;
                delete[] p_;
                delete[] rop_;
                delete[] pp_;
                delete[] u_;
                ro_ = new double[kl];
                p_ = new double[kl];
                u_ = new double[kl];
                rop_ = new double[kl];
                pp_ = new double[kl];
            }
        }

        takeOmni << <1, 1 >> > (dev_ro2, dev_p2, dev_ro_p2, dev_p_p2, dev_u2, dev_Ro_Omni, dev_P_Omni, dev_U_Omni, dev_Time_Omni, dev_T_all, dev_mas_Omni);
        takeVoyadger << <1, 1 >> > (dev_T_do, dev_T_all, dev_Time_V, dev_mas_V, dev_voy);
        
        // Launch a kernel on the GPU with one thread for each element.
        //calcul << < (int)(1.0 * *NN / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (dev_ro2, dev_u2, dev_p2, dev_ro, dev_u, dev_p,//
        //    dev_T, dev_T_do, dev_N);

        calcul_component << < (int)(1.0 * *NN / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (dev_ro2, dev_u2, dev_p2, dev_ro, dev_u, dev_p,//
            dev_ro_p2, dev_ro_p, dev_p_p2, dev_p_p, dev_T, dev_T_do, dev_N);

        funk_time << <1, 1 >> > (dev_T, dev_T_do, dev_T_all);

        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            goto Error;
        }

        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
            goto Error;
        }

        cudaStatus = hipMemcpy(voy, dev_voy, sizeof(int), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMemcpy failed!");
            goto Error;
        }

        // ����� ���������� ������ ��������
        if (*voy == 1)
        {
            *voy == 0;
            cudaStatus = hipMemcpy(dev_voy, voy, sizeof(int), hipMemcpyHostToDevice);
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "hipMemcpy failed!");
                goto Error;
            }


            cudaStatus = hipMemcpy(mas_V, dev_mas_V, sizeof(int), hipMemcpyDeviceToHost);
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "hipMemcpy failed!");
                goto Error;
            }

            takeVoyadger2 << <1, 1 >> > (dev_ro, dev_ro_p, dev_p_p, dev_u, dev_Dist_V, dev_mas_V, dev_j1, dev_j2, dev_j3); // ���� 2 �����-��
            cudaStatus = hipMemcpy(j1, dev_j1, sizeof(double), hipMemcpyDeviceToHost);
            cudaStatus = hipMemcpy(j2, dev_j2, sizeof(double), hipMemcpyDeviceToHost);
            cudaStatus = hipMemcpy(j3, dev_j3, sizeof(double), hipMemcpyDeviceToHost);
            cudaStatus = hipMemcpy(T_all, dev_T_all, sizeof(double), hipMemcpyDeviceToHost);
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "hipMemcpy failed 7!");
                goto Error;
            }
            fout5 << *T_all << " " << Dist_V[*mas_V] << " " << *j1 * 7.0 << " " << *j2 * 375.0 << " " << *j3 / (2.0 * *j1 * 5.86922 * 0.00000001) << //
                " " << Ro_V[*mas_V] << " " << U_V[*mas_V] << " " << P_V[*mas_V] << endl;
        }

        if (step % 10000 == 0)
        {
            cudaStatus = hipMemcpy(T_all, dev_T_all, sizeof(double), hipMemcpyDeviceToHost);
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "hipMemcpy failed 8!");
                goto Error;
            }
            cudaStatus = hipMemcpy(mas_Omni, dev_mas_Omni, sizeof(int), hipMemcpyDeviceToHost);
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "hipMemcpy failed 9!");
                goto Error;
            }
            cout << *T_all << " " << *mas_Omni << " " << *mas_V << endl;

            // ����� ��������� ���������� �������� ����
            if (*mas_Omni > 0.85 * N_O1)
            {
                N_O2 += (*mas_Omni - 1);
                takeDataOmni(Ro_Omni, P_Omni, U_Omni, Time_Omni, N_O1, N_O2);
                cudaStatus = hipMemcpy(dev_Time_Omni, Time_Omni, N_O1 * sizeof(double), hipMemcpyHostToDevice);
                if (cudaStatus != hipSuccess) {
                    fprintf(stderr, "hipMemcpy failed 10!");
                    goto Error;
                }
                cudaStatus = hipMemcpy(dev_U_Omni, U_Omni, N_O1 * sizeof(double), hipMemcpyHostToDevice);
                if (cudaStatus != hipSuccess) {
                    fprintf(stderr, "hipMemcpy failed 11!");
                    goto Error;
                }
                cudaStatus = hipMemcpy(dev_Ro_Omni, Ro_Omni, N_O1 * sizeof(double), hipMemcpyHostToDevice);
                if (cudaStatus != hipSuccess) {
                    fprintf(stderr, "hipMemcpy failed 12!");
                    goto Error;
                }
                cudaStatus = hipMemcpy(dev_P_Omni, P_Omni, N_O1 * sizeof(double), hipMemcpyHostToDevice);
                if (cudaStatus != hipSuccess) {
                    fprintf(stderr, "hipMemcpy failed!");
                    goto Error;
                }
                *mas_Omni = 0;
                cudaStatus = hipMemcpy(dev_mas_Omni, mas_Omni, sizeof(int), hipMemcpyHostToDevice);
                if (cudaStatus != hipSuccess) {
                    fprintf(stderr, "hipMemcpy failed!");
                    goto Error;
                }
                cout << "Refrishing the Omny's arrays" << endl;
            }
        }


        // ������ �������� 
        if (step % 1000000 == 0)
        {
            cudaStatus = hipMemcpy(T_all, dev_T_all, sizeof(double), hipMemcpyDeviceToHost);
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "hipMemcpy failed 8!");
                goto Error;
            }

            cudaStatus = hipMemcpy(ro, dev_ro, *NN * sizeof(double), hipMemcpyDeviceToHost);
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "hipMemcpy failed!");
                goto Error;
            }

            cudaStatus = hipMemcpy(p, dev_p, *NN * sizeof(double), hipMemcpyDeviceToHost);
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "hipMemcpy failed!");
                goto Error;
            }

            cudaStatus = hipMemcpy(ro2, dev_ro_p, *NN * sizeof(double), hipMemcpyDeviceToHost);
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "hipMemcpy failed!");
                goto Error;
            }

            cudaStatus = hipMemcpy(p2, dev_p_p, *NN * sizeof(double), hipMemcpyDeviceToHost);
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "hipMemcpy failed!");
                goto Error;
            }

            cudaStatus = hipMemcpy(u, dev_u, *NN * sizeof(double), hipMemcpyDeviceToHost);
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "hipMemcpy failed!");
                goto Error;
            }

            fout2.open("save.txt");
            fout2 << *T_all << " " << *NN << endl;
            for (int i = 0; i < *NN; i++)
            {
                double r = L + i * dx;
                fout2 << r << " " << ro[i] << " " << p[i] << " " << u[i] << " " << ro2[i] << " " << p2[i] << endl;
                //fout2 << r << " " << (ro[i] - ro2[i]) * 7.0 << " " << p[i] << " " << u[i] * 375.0 << " " << p[i] / (2.0 * (ro[i] - ro2[i]) * 5.86922 * 0.00000001) << endl;
            }
            fout2.close();

        }
    }


    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(ro, dev_ro, *NN * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(p, dev_p, *NN * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(ro2, dev_ro_p, *NN * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(p2, dev_p_p, *NN * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(u, dev_u, *NN * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    
    fout2.open("save_2.txt");
    fout2 << *T_all << " " << *NN << endl;
    for (int i = 0; i < *NN; i++)
    {
        double r = L + i * dx;
        fout2 << r << " " << ro[i] << " " << p[i] << " " << u[i] << " " << ro2[i] << " " << p2[i] << endl;
        //fout2 << r << " " << (ro2[i] - ro[i])  * 7.0 << " " << p[i] << " " << u[i] * 375.0 << " " << p[i]/ (2.0 * (ro2[i] - ro[i]) * 5.86922 * 0.00000001) << " " << p2[i] / (2.0 * ro2[i] * 5.86922 * 0.00000001) <<  endl;
    }
    fout2.close();

Error:
    
    return cudaStatus;
}
